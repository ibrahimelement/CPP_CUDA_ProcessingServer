#include "hip/hip_runtime.h"
#include "CudaHeader.cuh"

#include <iostream>
#include <string>
#include <stdio.h>
#include <mutex>
#include <atomic>

#include "hip/hip_runtime.h"
#include ""

#include "SHA256.cuh"
#include "json11.h"

__device__ bool checkZeroPadding(unsigned char* sha, uint8_t difficulty) {

	bool isOdd = difficulty % 2 != 0;
	uint8_t max = (difficulty / 2) + 1;

	/*
		Odd : 00 00 01 need to check 0 -> 2
		Even : 00 00 00 1 need to check 0 -> 3
		odd : 5 / 2 = 2 => 2 + 1 = 3
		even : 6 / 2 = 3 => 3 + 1 = 4
	*/
	for (uint8_t cur_byte = 0; cur_byte < max; ++cur_byte) {
		uint8_t b = sha[cur_byte];
		if (cur_byte < max - 1) { // Before the last byte should be all zero
			if (b != 0) return false;
		}
		else if (isOdd) {
			if (b > 0x0F || b == 0) return false;
		}
		else if (b <= 0x0f) return false;

	}

	return true;

}

__device__ uint8_t nonce_to_str(uint64_t nonce, unsigned char* out) {
	uint64_t result = nonce;
	uint8_t remainder;
	uint8_t nonce_size = nonce == 0 ? 1 : floor(log10((double)nonce)) + 1;
	uint8_t i = nonce_size;
	while (result >= 10) {
		remainder = result % 10;
		result /= 10;
		out[--i] = remainder + '0';
	}

	out[0] = result + '0';
	i = nonce_size;
	out[i] = 0;
	return i;
}

void pre_sha256() {
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}

__global__ void HashKernel(unsigned long int hashTotal, BYTE* hashInput, unsigned int numZero, unsigned int inputSize, BYTE* hashOutput, BYTE* nonceContainer, unsigned int* hashIndex, unsigned int* found)
{

	unsigned long int totalHashed = 0;
	unsigned char nonceTemp[12];

	for (unsigned long int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < hashTotal && !(*found);
		i += blockDim.x * gridDim.x)
	{

		unsigned long int nonceContainerOffset = (i * 6);
		unsigned long int hashContainerOffset = (i * 32);

		size_t nonceSize = nonce_to_str(i, nonceTemp);

		totalHashed++;
		SHA256_CTX ctx;
		sha256_init(&ctx);
		sha256_update(&ctx, hashInput, inputSize);
		sha256_update(&ctx, nonceTemp, nonceSize);
		sha256_final(&ctx, hashOutput + hashContainerOffset);

		if (checkZeroPadding(hashOutput + hashContainerOffset, numZero) && atomicExch(found, 1) == 0) {
			*hashIndex = i;
			printf("FOUND! %d %d, hashOutput location: %d, nonce output location: %d\n", numZero, *hashIndex, hashContainerOffset, nonceContainerOffset);
		}

	}

}

CudaHashContext::CudaHashContext(){}
CudaHashContext::~CudaHashContext(){

	std::cout << "Deallocating resources" << std::endl;

	// Free device memory
	if (this->_dev_outFoundAtomic != nullptr) {
		hipFree(this->_dev_outFoundAtomic);
	}
	if (this->_dev_outFoundIndex != nullptr) {
		hipFree(this->_dev_outFoundIndex);
	}
	if (this->_dev_hashOutput != nullptr) {
		hipFree(this->_dev_hashOutput);
	}
	if (this->_dev_nonceContainer != nullptr) {
		hipFree(this->_dev_nonceContainer);
	}
	if (this->_dev_hashInput != nullptr) {
		hipFree(this->_dev_hashInput);
	}

	// Free host memory
	if (this->_localHashOutput != nullptr) {
		delete this->_localHashOutput;
	}

	this->_ResetDevice();

}

bool CudaHashContext::_ResetDevice() {

	std::cout << "Resetting device" << std::endl;
	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return true;
	}

	return false;
}

bool CudaHashContext::_AllocateResources() {

	try {

		hipError_t cudaStatus;
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			throw std::exception("Failed to allocated resources");
		}

		cudaStatus = hipMalloc((void**)&this->_dev_outFoundAtomic, sizeof(unsigned int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw std::exception("Failed to allocate resources");
		}

		cudaStatus = hipMalloc((void**)&this->_dev_outFoundIndex, sizeof(unsigned int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw std::exception("Failed to allocate resources");
		}

		cudaStatus = hipMalloc((void**)&this->_dev_nonceContainer, sizeof(unsigned char) * this->HASH_COUNT * 6);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw std::exception("Failed to allocate resources");
		}

		cudaStatus = hipMalloc((void**)&this->_dev_hashOutput, this->HASH_COUNT * sizeof(BYTE) * 32);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw std::exception("Failed to allocate resources");
		}

		cudaStatus = hipMalloc((void**)&this->_dev_hashInput, this->INPUT_SIZE);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw std::exception("Failed to allocate resources");
		}

		cudaStatus = hipMalloc((void**)&this->_dev_hashInputSize, sizeof(unsigned int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			throw std::exception("Failed to allocate resources");
		}

		pre_sha256();

		return true;

	}
	catch (std::exception err) {
		std::cout << "Critical error allocating resources: " << err.what() << std::endl;
	}
	
	return false;
}


void CudaHashContext::Initialize() {
	bool hasAllocated = this->_AllocateResources();
}

CudaHashContext::HashChallenge CudaHashContext::_ExecuteChallenge() {

	hipError_t cudaStatus;

	HashKernel<<<1024, 32>>>(
		this->HASH_COUNT,
		this->_dev_hashInput,
		this->numZeros,
		this->strHashInput.size(),
		this->_dev_hashOutput,
		this->_dev_nonceContainer,
		this->_dev_outFoundIndex,
		this->_dev_outFoundAtomic
	);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		throw std::exception("Launched failed");
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		throw std::exception("Failed to syncronize with device");
	}

	// Copy output values
	HashChallenge hashRes;
	unsigned char* hostDigestSolution = new unsigned char[32];
	unsigned int* hasFoundSolution = new unsigned int;
	unsigned int* solutionIndex = new unsigned int;

	hipMemcpy(
		hasFoundSolution,
		this->_dev_outFoundAtomic,
		sizeof(unsigned int),
		hipMemcpyDeviceToHost
	);

	if (*hasFoundSolution) {
		
		hashRes.success = true;
		
		hipMemcpy(
			solutionIndex,
			this->_dev_outFoundIndex,
			sizeof(unsigned int),
			hipMemcpyDeviceToHost
		);

		hipMemcpy(
			hostDigestSolution,
			this->_dev_hashOutput + (32 * (*solutionIndex)),
			32,
			hipMemcpyDeviceToHost
		);

		hashRes.correctOutput = hash_to_string(
			hostDigestSolution
		);

		hashRes.numIterations = *solutionIndex;

	}

	delete[] hostDigestSolution;
	delete solutionIndex;
	delete hasFoundSolution;

	return hashRes;

}

CudaHashContext::HashChallenge CudaHashContext::SolveChallenge(std::string input, unsigned int zeros) {

	// Clear buffers from prior runs
	hipMemset(
		this->_dev_hashInput,
		0,
		this->INPUT_SIZE
	);
	hipMemset(
		this->_dev_outFoundIndex,
		0,
		sizeof(unsigned int)
	);
	hipMemset(
		this->_dev_outFoundAtomic,
		0,
		sizeof(unsigned int)
	);


	// Copy over new input data
	hipMemcpy(
		this->_dev_hashInput,
		input.c_str(),
		input.size(),
		hipMemcpyHostToDevice
	);
	hipMemset(
		this->_dev_hashInputSize,
		input.length(),
		sizeof(unsigned int)
	);

	this->strHashInput = input;
	this->numZeros = zeros;

	HashChallenge res = this->_ExecuteChallenge();

	return res;
}

std::string CudaHashContext::HashChallenge::Serialize() {

	json11::Json obj = json11::Json::object({
		{"success", this->success},
		{"output", this->correctOutput},
		{"postfix", (int)this->numIterations}
	});

	return obj.dump();

}